#include "hip/hip_runtime.h"
//CUDA version of main

#include <cstdio>
#include <fstream>
#include <iostream>

#include "vec3.h"

#define checkCudaErrors(val) check_cuda( (val), #val, __FILE__, __LINE__ )
void check_cuda(hipError_t result, char const *const func, const char *const file, int const line) {
    if (result) {
        std::cerr << "CUDA error = " << static_cast<unsigned int>(result) << " at " <<
        file << ":" << line << " '" << func << "' \n";
        // Make sure we call CUDA Device Reset before exiting
        hipDeviceReset();
        exit(99);
    }
}

__global__ void render(vec3* fb_vec3, int* max_x, int* max_y){
  int i = threadIdx.x + blockDim.x * blockIdx.x;
  int j = threadIdx.y + blockDim.y * blockIdx.y;
  if (i >= *max_x || j >= *max_y) return;
  int pixelIndex = j* (*max_x) + i;
  double r = (double)i/(*max_x -1);
  double g = (double)j/(*max_y -1);
  double b = 0.25;
  fb_vec3[pixelIndex] = vec3(r,g,b);
}

int main(){

  int tx = 8;
  int ty = 8;

  //Allocate memory on CPU
  int *nx_cpu = new int(256);
  int *ny_cpu = new int(256);
  float *fb_cpu = new float;

  //allocates memory on the GPU, first argument is a pointer to a pointer to that memory
  int num_pixels = *nx_cpu * (*ny_cpu);

  //allocates memory on the CPU
  size_t fb_vec3_size = num_pixels*sizeof(vec3);
  vec3* fb_vec3_cpu = (vec3*)malloc(fb_vec3_size);

  //allocates memory on the GPU, first argument is a pointer to a pointer to that memory
  vec3* fb_vec3_gpu;
  checkCudaErrors(hipMalloc((void **)&fb_vec3_gpu, fb_vec3_size));

  int *nx_gpu;
  checkCudaErrors(hipMalloc((void**)&nx_gpu, sizeof(int)));
  checkCudaErrors(hipMemcpy(nx_gpu, nx_cpu, sizeof(int),hipMemcpyHostToDevice));

  int *ny_gpu;
  checkCudaErrors(hipMalloc((void**)&ny_gpu, sizeof(int)));
  checkCudaErrors(hipMemcpy(ny_gpu, ny_cpu, sizeof(int),hipMemcpyHostToDevice));

  dim3 blocks(*nx_cpu/tx+1,*ny_cpu/ty+1);
  dim3 threads(tx,ty);
  render<<<blocks,threads>>>(fb_vec3_gpu,nx_gpu,ny_gpu);
  hipDeviceSynchronize();
  checkCudaErrors(hipMemcpy(fb_vec3_cpu,fb_vec3_gpu,fb_vec3_size,hipMemcpyDeviceToHost));

  std::cout << "P3\n" << *nx_cpu << ' ' << *ny_cpu << "\n255\n";

  for (int j = *ny_cpu-1; j >= 0; --j) {
    for (int i = 0; i < *nx_cpu; ++i) {
      int pixelIndex = j* (*nx_cpu) + i;
      vec3 pixel = fb_vec3_cpu[pixelIndex];
      int ir = static_cast<int>(255.999 * pixel.x());
      int ig = static_cast<int>(255.999 * pixel.y());
      int ib = static_cast<int>(255.999 * pixel.z());
      std::cout << ir << ' ' << ig << ' ' << ib << '\n';
    }
  }

  //Clean memory on CPU
  delete nx_cpu;
  delete ny_cpu;
  delete fb_cpu;

  return 1;
}
