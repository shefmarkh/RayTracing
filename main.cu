#include "hip/hip_runtime.h"
//CUDA version of main

#include <cstdio>
#include <fstream>
#include <iostream>

#include "vec3.h"
#include "ray.h"

#define checkCudaErrors(val) check_cuda( (val), #val, __FILE__, __LINE__ )
void check_cuda(hipError_t result, char const *const func, const char *const file, int const line) {
    if (result) {
        std::cerr << "CUDA error = " << static_cast<unsigned int>(result) << " at " <<
        file << ":" << line << " '" << func << "' \n";
        // Make sure we call CUDA Device Reset before exiting
        hipDeviceReset();
        exit(99);
    }
}

__device__ color ray_color(const ray& r) {
  vec3 unit_direction = unit_vector(r.direction());
  double t = 0.5*(unit_direction.y() + 1.0);
  return (1.0-t)*color(1.0, 1.0, 1.0) + t*color(0.5, 0.7, 1.0);
}


__global__ void render(color* fb_color, int* max_x, int* max_y, double *aspect_ratio){
  int i = threadIdx.x + blockDim.x * blockIdx.x;
  int j = threadIdx.y + blockDim.y * blockIdx.y;
  if (i >= *max_x || j >= *max_y) return;
  int pixelIndex = j* (*max_x) + i;

  double viewport_height = 2.0;
  double viewport_width = *aspect_ratio * viewport_height;
  double focal_length = 1.0;

  point3 origin = point3(0,0,0);
  vec3 horizontal = vec3(viewport_width, 0, 0);
  vec3 vertical = vec3(0, viewport_height, 0);
  vec3 lower_left_corner = origin - horizontal/2 - vertical/2 - vec3(0, 0, focal_length);

  double u = double(i) / (*max_x-1);
  double v = double(j) / (*max_y-1);
  ray theRay(origin, lower_left_corner + u*horizontal + v*vertical - origin);
  fb_color[pixelIndex] = ray_color(theRay);

}

int main(){

  int tx = 8;
  int ty = 8;

  //Allocate memory on CPU
  const double aspect_ratio_cpu = 16.0 / 9.0;
  int nx_cpu = 384;
  int ny_cpu = nx_cpu/aspect_ratio_cpu;

  int num_pixels = nx_cpu * ny_cpu;
  color fb_color_cpu[num_pixels];

  //allocates memory on the GPU, first argument is a pointer to a pointer to that memory
  size_t fb_color_size = num_pixels*sizeof(color);
  color* fb_color_gpu;
  checkCudaErrors(hipMalloc((void **)&fb_color_gpu, fb_color_size));

  int *nx_gpu;
  checkCudaErrors(hipMalloc((void**)&nx_gpu, sizeof(int)));
  checkCudaErrors(hipMemcpy(nx_gpu, &nx_cpu, sizeof(int),hipMemcpyHostToDevice));

  int *ny_gpu;
  checkCudaErrors(hipMalloc((void**)&ny_gpu, sizeof(int)));
  checkCudaErrors(hipMemcpy(ny_gpu, &ny_cpu, sizeof(int),hipMemcpyHostToDevice));

  double *aspect_ratio_gpu;
  checkCudaErrors(hipMalloc((void**)&aspect_ratio_gpu, sizeof(double)));
  checkCudaErrors(hipMemcpy(aspect_ratio_gpu, &aspect_ratio_cpu, sizeof(double),hipMemcpyHostToDevice));

  dim3 blocks(nx_cpu/tx+1,ny_cpu/ty+1);
  dim3 threads(tx,ty);
  render<<<blocks,threads>>>(fb_color_gpu,nx_gpu,ny_gpu,aspect_ratio_gpu);
  hipDeviceSynchronize();
  checkCudaErrors(hipMemcpy(&fb_color_cpu,fb_color_gpu,fb_color_size,hipMemcpyDeviceToHost));

  std::cout << "P3\n" << nx_cpu << ' ' << ny_cpu << "\n255\n";

  for (int j = ny_cpu-1; j >= 0; --j) {
    for (int i = 0; i < nx_cpu; ++i) {
      int pixelIndex = j* (nx_cpu) + i;
      color pixel = fb_color_cpu[pixelIndex];
      int ir = static_cast<int>(255.999 * pixel.x());
      int ig = static_cast<int>(255.999 * pixel.y());
      int ib = static_cast<int>(255.999 * pixel.z());
      std::cout << ir << ' ' << ig << ' ' << ib << '\n';
    }
  }

  //Clean memory on GPU
  checkCudaErrors(hipFree(nx_gpu));
  checkCudaErrors(hipFree(ny_gpu));
  checkCudaErrors(hipFree(fb_color_gpu));

  return 1;
}
