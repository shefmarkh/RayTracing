#include "hip/hip_runtime.h"
//CUDA version of main

#include <cstdio>
#include <fstream>
#include <iostream>

#include "vec3.h"
#include "ray.h"

#define checkCudaErrors(val) check_cuda( (val), #val, __FILE__, __LINE__ )
void check_cuda(hipError_t result, char const *const func, const char *const file, int const line) {
    if (result) {
        std::cerr << "CUDA error = " << static_cast<unsigned int>(result) << " at " <<
        file << ":" << line << " '" << func << "' \n";
        // Make sure we call CUDA Device Reset before exiting
        hipDeviceReset();
        exit(99);
    }
}

__device__ color ray_color(const ray& r) {
  vec3 unit_direction = unit_vector(r.direction());
  double t = 0.5*(unit_direction.y() + 1.0);
  return (1.0-t)*color(1.0, 1.0, 1.0) + t*color(0.5, 0.7, 1.0);
}


__global__ void render(color* fb_color, int* max_x, int* max_y){
  int i = threadIdx.x + blockDim.x * blockIdx.x;
  int j = threadIdx.y + blockDim.y * blockIdx.y;
  if (i >= *max_x || j >= *max_y) return;
  int pixelIndex = j* (*max_x) + i;
  double r = (double)i/(*max_x -1);
  double g = (double)j/(*max_y -1);
  double b = 0.25;
  fb_color[pixelIndex] = color(r,g,b);
}

int main(){

  int tx = 8;
  int ty = 8;

  //Allocate memory on CPU
  int *nx_cpu = new int(256);
  int *ny_cpu = new int(256);
  float *fb_cpu = new float;

  //allocates memory on the GPU, first argument is a pointer to a pointer to that memory
  int num_pixels = *nx_cpu * (*ny_cpu);

  //allocates memory on the CPU
  size_t fb_color_size = num_pixels*sizeof(color);
  color* fb_color_cpu = (color*)malloc(fb_color_size);

  //allocates memory on the GPU, first argument is a pointer to a pointer to that memory
  color* fb_color_gpu;
  checkCudaErrors(hipMalloc((void **)&fb_color_gpu, fb_color_size));

  int *nx_gpu;
  checkCudaErrors(hipMalloc((void**)&nx_gpu, sizeof(int)));
  checkCudaErrors(hipMemcpy(nx_gpu, nx_cpu, sizeof(int),hipMemcpyHostToDevice));

  int *ny_gpu;
  checkCudaErrors(hipMalloc((void**)&ny_gpu, sizeof(int)));
  checkCudaErrors(hipMemcpy(ny_gpu, ny_cpu, sizeof(int),hipMemcpyHostToDevice));

  dim3 blocks(*nx_cpu/tx+1,*ny_cpu/ty+1);
  dim3 threads(tx,ty);
  render<<<blocks,threads>>>(fb_color_gpu,nx_gpu,ny_gpu);
  hipDeviceSynchronize();
  checkCudaErrors(hipMemcpy(fb_color_cpu,fb_color_gpu,fb_color_size,hipMemcpyDeviceToHost));

  std::cout << "P3\n" << *nx_cpu << ' ' << *ny_cpu << "\n255\n";

  for (int j = *ny_cpu-1; j >= 0; --j) {
    for (int i = 0; i < *nx_cpu; ++i) {
      int pixelIndex = j* (*nx_cpu) + i;
      color pixel = fb_color_cpu[pixelIndex];
      int ir = static_cast<int>(255.999 * pixel.x());
      int ig = static_cast<int>(255.999 * pixel.y());
      int ib = static_cast<int>(255.999 * pixel.z());
      std::cout << ir << ' ' << ig << ' ' << ib << '\n';
    }
  }

  //Clean memory on CPU
  delete nx_cpu;
  delete ny_cpu;
  delete fb_cpu;

  return 1;
}
